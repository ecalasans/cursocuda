//
// Created by eric on 22/12/2020.
//

#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>
#include <ctime>
#include <sys/time.h>

#define N 512
#define TAM_BLOCO 8

__global__ void deviceAdd(int* a, int* b, int* c){
    int indice = threadIdx.x + blockIdx.x * blockDim.x;
    c[indice] = a[indice] + b[indice];
}

void fillArray(int* data){
    for (int idx = 0; idx < N; idx++){
        data[idx] = idx;
    }
}

void printResults(int* a, int* b, int* c){
    for (int idx = 0; idx < N; idx++){
        printf("A[%d] + B[%d] = %d\n", a[idx], b[idx], c[idx]);
    }
}

int main(){
    int *a, *b, *c;  // apontam para a device memory(global)
    int *d_a, *d_b, *d_c;
    struct timeval inicio, fim;

    int size = N * sizeof(int);
    std::size_t tam = N * sizeof(int);
    int blocos = N / TAM_BLOCO;
    std::cout << blocos << std::endl;

    a = (int *)malloc(size);
    fillArray(a);

    b = (int *)malloc(size);
    fillArray(b);

    c = (int *)malloc(size);

    hipMalloc((void **)&d_a, tam);
    hipMalloc((void **)&d_b, tam);
    hipMalloc((void **)&d_c, tam);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    gettimeofday(&inicio, NULL);
    deviceAdd<<<blocos,TAM_BLOCO>>>(d_a, d_b, d_c);   //<<<blocos, threads>>>
    gettimeofday(&fim, NULL);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printResults(a, b, c);

    int diferenca = fim.tv_usec - inicio.tv_usec;
    printf("Tempo de execucao = %d\n", diferenca);

    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
