
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>

__global__ void printFromGPU(){
    printf("Olá!  Sou a thread [%d, %d] falando da GPU!\n",
           threadIdx.x, blockIdx.x);
    printf("\n");
}


int main(){
    system("clear");
    std::cout << "Olá!  Sou a CPU!!" << std::endl;
    printFromGPU<<<1,1>>>();  // 1 thread 1 bloco

    printFromGPU<<<2,1>>>();  // 2 threads em 2 blocos

    printFromGPU<<<1,2>>>();  // 2 threads no mesmo bloco
    hipDeviceSynchronize();

    return 0;
}

