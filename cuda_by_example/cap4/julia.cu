#include "hip/hip_runtime.h"
//
// Created by eric on 18/01/2021.
//

#include "../exemplos/common/book.h"
#include "../exemplos/common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex{
    float r;
    float i;

    hipComplex(float a, float b) : r(a), i(b) {};

    __device__ float magnitude2(){
        return r*r + i*i;
    }

    __device__ hipComplex operator*(const hipComplex& a){
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }

    __device__ hipComplex operator+(const hipComplex& a){
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y, hipComplex& semente){
    const float escala = 1.5;

    float jx = escala * (float)(DIM/2 - x)/(DIM/2);
    float jx = escala * (float)(DIM/2 - y)(DIM/2);

    hipComplex a(jx, jy);

    for (int i=0; i < 200; ++i){
        a = a*a + semente;

        if(a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel(unsigned char* ptr, hipComplex& semente){
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y*gridDim.x;

    int julia_valor = julia(x, y, semente);

    ptr[offset*4 + 0] = 255*julia_valor;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;

}


