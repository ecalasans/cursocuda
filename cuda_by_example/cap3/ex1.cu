//
// Created by eric on 31/12/2020.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

__global__ void addVector(int a, int b, int* c){
    *c = a + b;
}

int main(){
    int c = 0; // no host
    int count = 0;
    int* dev_c;  // no device

    hipDeviceProp_t prop;

    hipMalloc((void**)&dev_c, sizeof(int)); // aloca memória no device

    addVector<<<1,1>>>(2, 7, dev_c);  // <<<bloco, thread>>>(parâmetros da função)

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);  // copia do device pro host

    printf("Resultado:  %d\n", c);

    hipFree(dev_c); // libera memória do device

    hipGetDeviceCount(&count);

    printf("Total de dispositivos graficos = %d\n", count);

    //Mostra propriedades da GPU
    hipGetDeviceProperties(&prop, 0); //Parâmetros:  variável contendo propriedades e nº do dispositivo

    printf("Nome do dispositivo:  %s\n", prop.name);
    return 0;
}