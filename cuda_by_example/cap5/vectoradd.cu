//
// Created by eric on 21/01/2021.
//

#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>
#include <ctime>
#include <sys/time.h>

#define N 512
#define TAM_BLOCO 8

__global__ void deviceAdd(int* a, int* b, int* c){
    int indice = threadIdx.x + blockIdx.x * blockDim.x;
    c[indice] = a[indice] + b[indice];
}

void fillArray(int* data){
    for (int idx = 0; idx < N; idx++){
        data[idx] = idx;
    }
}

void printResults(int* a, int* b, int* c){
    for (int idx = 0; idx < N; idx++){
        printf("A[%d] + B[%d] = %d\n", a[idx], b[idx], c[idx]);
    }
}

int main(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    struct timeval inicio, fim;

    int size = N * sizeof(int);
    std::size_t tam = N * sizeof(int);
    int n_blocos = N/TAM_BLOCO;

    a = (int *)malloc(size);
    fillArray(a);

    b = (int *)malloc(size);
    fillArray(b);

    c = (int *)malloc(size);

    hipMalloc((void **)&d_a, tam);
    hipMalloc((void **)&d_b, tam);
    hipMalloc((void **)&d_c, tam);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    gettimeofday(&inicio, NULL);
    deviceAdd<<<1,N>>>(d_a, d_b, d_c);   //<<<blocos, threads>>>
    gettimeofday(&fim, NULL);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printResults(a, b, c);

    int diferenca = fim.tv_usec - inicio.tv_usec;
    printf("Tempo de execucao = %d\n", diferenca);

    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}

