//
// Created by eric on 29/01/2021.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include "stdio.h"
#include <ctime>

#define imin(a, b) (a<b?a:b)

const int N = 33*1024;
const int THREADS_POR_BLOCO = 256;

__global__ void dot(float* a, float* b, float* c){
    __shared__ float cache[THREADS_POR_BLOCO];

    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int indice_cache = threadIdx.x;

    float temp = 0;

    while (tid < N){
        temp += a[tid] * b[tid];
        tid += blockDim.x*gridDim.x;
    }

    cache[indice_cache] += temp;

    //REDUÇÃO
    int i = blockIdx.x/2;

    while (i != 0){
        if (indice_cache < i){
            cache[indice_cache] += cache[indice_cache + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (indice_cache == 0){
        c[blockIdx.x] = cache[0];
    }
}

const int BLOCOS_POR_GRADE = imin(32, (N + THREADS_POR_BLOCO -1)/THREADS_POR_BLOCO);

int main(){
    float *a, *b, c, *parcial_c;
    float *dev_a, *dev_b, *dev_parcial_c;

    //ALOCAÇÃO
    a = new float[N];
    b = new float[N];
    parcial_c = new float[BLOCOS_POR_GRADE];

    hipMalloc((void**)&dev_a, N* sizeof(float));
    hipMalloc((void**)&dev_b, N* sizeof(float ));
    hipMalloc((void**)dev_parcial_c, BLOCOS_POR_GRADE* sizeof(float));

    //PREENCHE OS VETORES
    for (int i = 0; i < N; ++i){
        a[i] = i;
        b[i] = i*2;
    }

    //COPIA VETORES PARA A GPU
    hipMemcpy(dev_a, a, N* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N* sizeof(float), hipMemcpyHostToDevice);
    return 0;
}
