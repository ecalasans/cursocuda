//
// Created by eric on 22/12/2020.
//

#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>
#include <ctime>
#include <sys/time.h>

#define N 512

__global__ void deviceAdd(int* a, int* b, int* c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void fillArray(int* data){
    for (int idx = 0; idx < N; idx++){
        data[idx] = idx;
    }
}

void printResults(int* a, int* b, int* c){
    for (int idx = 0; idx < N; idx++){
        printf("A[%d] + B[%d] = %d\n", a[idx], b[idx], c[idx]);
    }
}

int main(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    struct timeval inicio, fim;

    int size = N * sizeof(int);
    std::size_t tam = N * sizeof(int);

    a = (int *)malloc(size);
    fillArray(a);

    b = (int *)malloc(size);
    fillArray(b);

    c = (int *)malloc(size);

    hipMalloc((void **)&d_a, tam);
    hipMalloc((void **)&d_b, tam);
    hipMalloc((void **)&d_c, tam);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    gettimeofday(&inicio, NULL);
    deviceAdd<<<N,1>>>(d_a, d_b, d_c);   //<<<blocos, threads>>>
    gettimeofday(&fim, NULL);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printResults(a, b, c);

    int diferenca = fim.tv_usec - inicio.tv_usec;
    printf("Tempo de execucao = %d\n", diferenca);

    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
