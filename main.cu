
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>

__global__ void printFromGPU(){
    printf("Olá!  Sou a thread [%d, %d] falando da GPU!\n",
           threadIdx.x, blockIdx.x);
}


int main(){

    std::cout << "Olá!  Sou a CPU!!" << std::endl;
    printFromGPU<<<1,1>>>();
    hipDeviceSynchronize();

    return 0;
}

